
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void VecAdd(float* A, float* B, float* C){
int i = threadIdx.x;
   if(i<5){C[i] = A[i] + B[i];}}
int main() {
 int n = 5;
 size_t size = n * sizeof(float);
 float *d_A, *d_B, *d_C;
 hipMalloc((void**)&d_A, size);
 hipMalloc((void**)&d_B, size);
 hipMalloc((void**)&d_C, size);
 float h_A[] = {1,2,3,4,5};
 float h_B[] = {10,20,30,40,50};
 float h_C[] = {0,0,0,0,0};
 hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
 hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
 int nThreadsPerBlock = 256;
 //int nBlocks = n / nThreadsPerBlock;
 VecAdd<<<1, nThreadsPerBlock>>>(d_A, d_B, d_C);
 hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
 for(int i =0;i<n;i++){printf("%f \n",h_C[i]);}
 hipFree(d_A); hipFree(d_B); hipFree(d_C);
}
