
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <time.h>
#include <stdlib.h>
__global__ void scalarValueMul(float * a, int n  , int  value)
{

int i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i<n)
  {
      a[i]= a[i]*value;
      printf("  blockDim.x = %d   blockIdx.x = %d   threadIdx.x = %d    i (%d)  a[i] = %.2f\n",blockDim.x,blockIdx.x,threadIdx.x,i,a[i]);
  }
  

}


void init (float * v, int n)
{
int i;
srand(time(NULL));
     for(i =0;i<n;i++)
    {
            v[i]=i;
    }
}

int main (int argc, char ** argv)
{
         
   int n,value;
   scanf(" %d",&n);   
   scanf(" %d",&value);        
   float * a=(float*) malloc( sizeof(float)*n);
   float * d_a;
   init(a,n);
   hipMalloc(&d_a, sizeof(float)*n);
   hipMemcpy(d_a,a,sizeof(float)*n,hipMemcpyHostToDevice);
   scalarValueMul<<<1,n>>>(d_a,n,value);
   hipMemcpy(a, d_a, sizeof(float)*n,hipMemcpyDeviceToHost);
        printf("\n");
        for(int i =0;i<n;i++)
        {
                printf("%.2f ",a[i]);
        }
        printf("\n");
return 0;
}