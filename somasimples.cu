


#include <hip/hip_runtime.h>
#include<stdio.h>
// Codigo a ser executado na GPU
__global__  void add(int a, int b, int * c)
{
*c = a+b; 
}


int main()
{
    int c;
    int *dev_c;
    int a,b;
    scanf(" %d %d",&a,&b);
    // aloca memoria na GPU
    hipMalloc((void**)&dev_c,sizeof(int));
    //chama o Kernel add (de forma assincrona). Chama  1 bloco com uma thread.
	add<<<1,1>>>(a,b,dev_c);
    // copia o resultado de volta para o host
    hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);
    printf("resultado da soma %d\n", c);
	return 0;
}
