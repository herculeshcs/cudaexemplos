
#include <hip/hip_runtime.h>
#include <cstdio>
__global__ void printHelloGPU()
{
	printf("Hello x: #%d  y: #%d\n", threadIdx.x, threadIdx.y);
}
int main()
{
    // threads organizadas em modo 2D dentro do bloco
	//dim3 threads(3,3);
	// 1 bloco com 3x3 threads
	printHelloGPU<<<1, 3>>>();
	hipDeviceSynchronize();
	return 0;
}
